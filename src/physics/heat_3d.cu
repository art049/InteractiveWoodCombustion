#include "hip/hip_runtime.h"
/* heat_3d.cu
 * 3-dim. Laplace eq. (heat eq.) by finite difference with shared memory
 * Ernest Yeung  ernestyalumni@gmail.com
 * 20160729
 */
#include "pressure.cuh"
#include "smoke_render.cuh"
#include "heat_3d.cuh"
#include "physics.h"
#include "vec3.cuh"

#define RAD 1 // radius of the stencil; helps to deal with "boundary conditions" at (thread) block's ends

__constant__ float dev_Deltat[1];

__constant__ float dev_heat_params[2];



int blocksNeeded( int N_i, int M_i) { return (N_i+M_i-1)/M_i; }

__device__ unsigned char clip(int n) { return n > 255 ? 255 : (n < 0 ? 0 : n);}

__device__ int idxClip( int idx, int idxMax) {
    return idx > (idxMax - 1) ? (idxMax - 1): (idx < 0 ? 0 : idx);
}

__device__ int flatten(int col, int row, int z, int width, int height, int depth) {
    return idxClip(col, width) + idxClip(row,height)*width + idxClip(z,depth)*width*height;
}
__device__ int flatten(int col, int row, int z) {
    return idxClip(col, GRID_COUNT) + idxClip(row,GRID_COUNT)*GRID_COUNT + idxClip(z,GRID_COUNT)*GRID_COUNT*GRID_COUNT;
}
__device__ float3 operator+(const float3 &a, const float3 &b) {
    return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
  }
__device__ float3 operator-(const float3 &a, const float3 &b) {
    return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);
  }
__device__ float3 operator*(const float3 &a, const float &b) {
  return make_float3(a.x*b, a.y*b, a.z*b);
}
__device__ float3 operator*(const float &b, const float3 &a) {
    return make_float3(a.x*b, a.y*b, a.z*b);
  }
__device__ int d_abs(int a) {
    return a > 0 ? a : -a;
}
__global__ void resetKernel(float * d_temp, float *  d_oldtemp,float3 *  d_vel,float3 *  d_oldvel,float *  d_smokedensity,float *  d_oldsmokedensity,  BC bc) {
    const int k_x = blockIdx.x*blockDim.x + threadIdx.x;
    const int k_y = blockIdx.y*blockDim.y + threadIdx.y;
    const int k_z = blockIdx.z*blockDim.z + threadIdx.z;

    if ((k_x >= dev_Ld[0]) || (k_y >= dev_Ld[1]) || (k_z >= dev_Ld[2])) return;
    const int k = flatten(k_x, k_y, k_z, dev_Ld[0], dev_Ld[1],dev_Ld[2]);
    d_temp[k] = d_oldtemp[k] = T_AMBIANT;
    d_vel[k] = d_oldvel[k] = {0.f, 0.f, 0.f};
    d_smokedensity[k] = d_oldsmokedensity[k] = 0.f;
    if(k_y < GRID_COUNT/6 && d_abs(k_z - GRID_COUNT/2) * d_abs(k_z - GRID_COUNT/2) + 
       d_abs(k_x - GRID_COUNT/2) * d_abs(k_x - GRID_COUNT/2) < GRID_COUNT  *GRID_COUNT / 25){
        d_smokedensity[k] = d_oldsmokedensity[k] =0.5f;
        d_temp[k] = d_oldtemp[k] = T_AMBIANT + 10.f;
    }
}


__device__ float3 getAlpham (float3 * d_vel, float3 pos, int k){
    // Iteratively compute alpha_m
    float3 alpha_m = d_vel[k] * dev_Deltat[0];
    for(uint i = 0; i < SEMILAGRANGIAN_ITERS; i++){
        float3 estimated = pos - alpha_m;
        if(estimated.x < 0) estimated.x = 0;
        if(estimated.y < 0) estimated.y = 0;
        if(estimated.z < 0) estimated.z = 0;
        uint3 b = {static_cast<uint>(estimated.x/BLOCK_SIZE),
                   static_cast<uint>(estimated.y/BLOCK_SIZE),
                   static_cast<uint>(estimated.z/BLOCK_SIZE)};
        float3 localCoord = (estimated - make_float3(b.x*BLOCK_SIZE, b.y*BLOCK_SIZE, b.z*BLOCK_SIZE)) * (1/BLOCK_SIZE); 
        alpha_m.x = (1-localCoord.x) * d_vel[flatten(b.x, b.y, b.z)  ].x+
                    localCoord.x     * d_vel[flatten(b.x+1, b.y, b.z)].x;
        alpha_m.y = (1-localCoord.y) * d_vel[flatten(b.x, b.y, b.z)  ].y+
                    localCoord.y     * d_vel[flatten(b.x, b.y+1, b.z)].y;
        alpha_m.z = (1-localCoord.z) * d_vel[flatten(b.x, b.y, b.z)  ].z+
                    localCoord.z     * d_vel[flatten(b.x, b.y, b.z+1)].z;
        alpha_m = alpha_m * dev_Deltat[0];
    }
    //CLIPPING ON FACES
    return alpha_m;
}
__device__ float3 fbuoyancy(float * d_smoke, float* d_temp, int k_x, int k_y, int k_z){
    const int k = flatten(k_x, k_y, k_z);
    float3 f = make_float3(0,0,0);
    f.y += -0.5 * BUOY_ALPHA*(d_smoke[k]+d_smoke[flatten(k_x,k_y+1,k_z)]);
    f.y += BUOY_BETA*((d_temp[k]+d_temp[flatten(k_x,k_y+1,k_z)]) * 0.5f - T_AMBIANT);
    return f;
}
__device__ float3 fconfinement(float3 * d_vorticity, int k_x, int k_y, int k_z){
    const int k = flatten(k_x, k_y, k_z);
    vec3 N(vec3(d_vorticity[flatten(k_x+1, k_y, k_z)]).length() - vec3(d_vorticity[k]).length(),
           vec3(d_vorticity[flatten(k_x, k_y+1, k_z)]).length() - vec3(d_vorticity[k]).length(),
           vec3(d_vorticity[flatten(k_x, k_y, k_z+1)]).length() - vec3(d_vorticity[k]).length());
    N /= BLOCK_SIZE; // NOT useful since we normalise
    //N.make_unit_vector();
    vec3 f = VORTICITY_EPSILON * BLOCK_SIZE * cross(N, vec3(d_vorticity[k]));
    return f.toFloat3();
}
__global__ void computeVorticity(float3 *d_vorticity, float3* d_vel, float3* d_ccvel){
    const int k_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int k_y = threadIdx.y + blockDim.y * blockIdx.y;
    const int k_z = threadIdx.z + blockDim.z * blockIdx.z;
    if ((k_x >= dev_Ld[0] ) || (k_y >= dev_Ld[1] ) || (k_z >= dev_Ld[2])) return;
    
    const int k = flatten(k_x, k_y, k_z, dev_Ld[0], dev_Ld[1],dev_Ld[2]);
    d_ccvel[k].x = (d_vel[k].x + d_vel[flatten(k_x+1, k_y, k_z)].x) * 0.5f;
    d_ccvel[k].y = (d_vel[k].y + d_vel[flatten(k_x, k_y+1, k_z)].y) * 0.5f;
    d_ccvel[k].x = (d_vel[k].z + d_vel[flatten(k_x, k_y, k_z+1)].z) * 0.5f;
    __syncthreads();
    d_vorticity[k].x = d_ccvel[flatten(k_x, k_y+1, k_z)].z - d_ccvel[flatten(k_x, k_y-1, k_z)].z - 
                       d_ccvel[flatten(k_x, k_y, k_z+1)].y + d_ccvel[flatten(k_x, k_y, k_z+1)].y;
    d_vorticity[k].x /= 2 * BLOCK_SIZE;
    d_vorticity[k].y = d_ccvel[flatten(k_x, k_y, k_z+1)].x - d_ccvel[flatten(k_x, k_y, k_z-1)].x - 
                       d_ccvel[flatten(k_x+1, k_y, k_z)].z + d_ccvel[flatten(k_x-1, k_y, k_z)].z;
    d_vorticity[k].y /= 2 * BLOCK_SIZE;
    d_vorticity[k].z = d_ccvel[flatten(k_x+1, k_y, k_z)].y - d_ccvel[flatten(k_x-1, k_y, k_z)].y - 
                       d_ccvel[flatten(k_x, k_y+1, k_z)].x + d_ccvel[flatten(k_x, k_y-1, k_z)].x;
    d_vorticity[k].z /= 2 * BLOCK_SIZE;
}
#include <stdio.h>

__global__ void velocityKernel(float *d_temp, float3* d_vel, float3* d_oldvel, float* d_smokedensity, float3* d_vorticity){
    const int k_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int k_y = threadIdx.y + blockDim.y * blockIdx.y;
    const int k_z = threadIdx.z + blockDim.z * blockIdx.z;
    if ((k_x >= dev_Ld[0] ) || (k_y >= dev_Ld[1] ) || (k_z >= dev_Ld[2])) return;
    const int k = flatten(k_x, k_y, k_z, dev_Ld[0], dev_Ld[1],dev_Ld[2]);
    if(k_x == 0 || k_x == GRID_COUNT - 1 || k_y == 0 || k_y == GRID_COUNT - 1 || k_z == 0 || k_z == GRID_COUNT - 1)
        return;

    // External forces
    float3 f = {0, 0, 0};
    float3 fext = {10,0,0};
    f = f + fext;
    f = f + fconfinement(d_vorticity, k_x, k_y, k_z);
    //printf("conf %f %f %f", fconf.x, fconf.y, fconf.z);
    f = f  + fbuoyancy(d_smokedensity, d_temp, k_x, k_y, k_z);
    d_vel[k] = d_oldvel[k] + f * dev_Deltat[0];
    
    // Semi Lagrangian Advection
    //float3 pos = make_float3(k_x*BLOCK_SIZE, k_y*BLOCK_SIZE, k_z*BLOCK_SIZE);
    float3 pos = make_float3((k_x+0.5f)*BLOCK_SIZE, (k_y+0.5f)*BLOCK_SIZE, (k_z+0.5f)*BLOCK_SIZE);

    float3 alpha_m = getAlpham(d_oldvel, pos, k);
    // Backtracing 
    float3 estimated = pos - 2 * alpha_m;
    if(estimated.x < 0) estimated.x = 0;
    if(estimated.y < 0) estimated.y = 0;
    if(estimated.z < 0) estimated.z = 0;
    uint3 b = {static_cast<uint>(estimated.x/BLOCK_SIZE),
               static_cast<uint>(estimated.y/BLOCK_SIZE),
               static_cast<uint>(estimated.z/BLOCK_SIZE)};
    float3 localCoord = (estimated - make_float3(b.x*BLOCK_SIZE, b.y*BLOCK_SIZE, b.z*BLOCK_SIZE)) * (1/BLOCK_SIZE);
    //Velocity per component
    float3 dv;
    dv.x = (1-localCoord.x) * d_oldvel[flatten(b.x, b.y, b.z)  ].x+
           localCoord.x     * d_oldvel[flatten(b.x+1, b.y, b.z)].x;
    dv.y = (1-localCoord.y) * d_oldvel[flatten(b.x, b.y, b.z)  ].y+
           localCoord.y     * d_oldvel[flatten(b.x, b.y+1, b.z)].y;
    dv.z = (1-localCoord.z) * d_oldvel[flatten(b.x, b.y, b.z)  ].z+
           localCoord.z     * d_oldvel[flatten(b.x, b.y, b.z+1)].z;
    dv = dv * 2 * dev_Deltat[0];
    d_vel[k] = d_vel[k] + dv;        
}
__device__ float scalarLinearInt(float* scalarField, float3 pos, float oobvalue){
    int x = static_cast<int> (pos.x / BLOCK_SIZE);
    int y = static_cast<int> (pos.y / BLOCK_SIZE);
    int z = static_cast<int> (pos.z / BLOCK_SIZE);
    //Getting voxel edges
    if(fabs(pos.x - x * BLOCK_SIZE) > fabs(pos.x - (x+1)*BLOCK_SIZE)) x++;
    if(fabs(pos.y - y * BLOCK_SIZE) > fabs(pos.y - (y+1)*BLOCK_SIZE)) y++;
    if(fabs(pos.z - z * BLOCK_SIZE) > fabs(pos.z - (z+1)*BLOCK_SIZE)) z++;
    //pos is inside voxels [x-1, x] [y-1, y] [z-1, z]
    //BOUND CHECK
    if(x <= 0 || x >= GRID_COUNT || y <= 0 || y >= GRID_COUNT || z <= 0 || z >= GRID_COUNT)
        return oobvalue;
    
    float tx = (pos.x /BLOCK_SIZE - (x - 0.5f) );
    float ty = (pos.y /BLOCK_SIZE - (y - 0.5f) );
    float tz = (pos.z /BLOCK_SIZE - (z - 0.5f) );

    // Bottom z then upper z
    float bybz = tx * scalarField[flatten(x,y-1,z-1)] + (1-tx) * scalarField[flatten(x-1,y-1,z-1)];
    float uybz = tx * scalarField[flatten(x,y,z-1)] + (1-tx) * scalarField[flatten(x-1,y,z-1)];
    float bz = (1-ty) * bybz + ty * uybz;
    float byuz = tx * scalarField[flatten(x,y-1,z)] + (1-tx) * scalarField[flatten(x-1,y-1,z)];
    float uyuz = tx * scalarField[flatten(x,y,z)] + (1-tx) * scalarField[flatten(x-1,y,z)];
    float uz = (1-ty) * byuz + ty * uyuz;
    return (1-tz) * bz + tz * uz;
}
__global__ void smokeAdvectionKernel(float *d_temp, float3* d_vel, float* d_smoke, float* d_oldsmoke){
    const int k_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int k_y = threadIdx.y + blockDim.y * blockIdx.y;
    const int k_z = threadIdx.z + blockDim.z * blockIdx.z;
    if ((k_x >= dev_Ld[0] ) || (k_y >= dev_Ld[1] ) || (k_z >= dev_Ld[2])) return;
    const int k = flatten(k_x, k_y, k_z, dev_Ld[0], dev_Ld[1],dev_Ld[2]);
    // Advection
    float3 pos = make_float3((k_x+0.5f)*BLOCK_SIZE, (k_y+0.5f)*BLOCK_SIZE, (k_z+0.5f)*BLOCK_SIZE);
    float3 alpha_m = getAlpham(d_vel, pos, k);
    // Backtracing 
    float3 estimated = pos - 2 * alpha_m;
    if(estimated.x < 0) estimated.x = 0;
    if(estimated.y < 0) estimated.y = 0;
    if(estimated.z < 0) estimated.z = 0;
    //uint3 b = {static_cast<uint>(estimated.x/BLOCK_SIZE),
    //           static_cast<uint>(estimated.y/BLOCK_SIZE),
    //           static_cast<uint>(estimated.z/BLOCK_SIZE)};
    //float3 localCoord = (estimated - make_float3(b.x*BLOCK_SIZE, b.y*BLOCK_SIZE, b.z*BLOCK_SIZE)) * (1 / BLOCK_SIZE);
    //float ds = d_smoke[flatten(b.x, b.y, b.z) ];

    float ds = scalarLinearInt(d_smoke, estimated, 0.f);
    ds = ds * 2 * dev_Deltat[0];
    //NEED OLD GRID FOR THIS
    d_smoke[k] = d_oldsmoke[k] + ds;
}
__global__ void tempAdvectionKernel(float *d_temp, float * d_oldtemp, float3* d_vel, float* d_smoke){
    const int k_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int k_y = threadIdx.y + blockDim.y * blockIdx.y;
    const int k_z = threadIdx.z + blockDim.z * blockIdx.z;
    if ((k_x >= dev_Ld[0] ) || (k_y >= dev_Ld[1] ) || (k_z >= dev_Ld[2])) return;
    const int k = flatten(k_x, k_y, k_z, dev_Ld[0], dev_Ld[1],dev_Ld[2]);
    // Advection
    float3 pos = make_float3((k_x+0.5f)*BLOCK_SIZE, (k_y+0.5f)*BLOCK_SIZE, (k_z+0.5f)*BLOCK_SIZE);
    float3 alpha_m = getAlpham(d_vel, pos, k);
    // Backtracing 
    float3 estimated = pos - 2 * alpha_m;
    if(estimated.x < 0) estimated.x = 0;
    if(estimated.y < 0) estimated.y = 0;
    if(estimated.z < 0) estimated.z = 0;
    //uint3 b = {static_cast<uint>(estimated.x/BLOCK_SIZE),
    //           static_cast<uint>(estimated.y/BLOCK_SIZE),
    //           static_cast<uint>(estimated.z/BLOCK_SIZE)};
    //float3 localCoord = (estimated - make_float3(b.x*BLOCK_SIZE, b.y*BLOCK_SIZE, b.z*BLOCK_SIZE)) * (1 / BLOCK_SIZE);
    //float ds = d_smoke[flatten(b.x, b.y, b.z) ];

    float dt = scalarLinearInt(d_temp, estimated, T_AMBIANT);
    dt = dt * 2 * dev_Deltat[0];
    __syncthreads();
    d_temp[k] = d_oldtemp[k] + dt;
}







void kernelLauncher(uchar4 *d_out,
                    float *d_temp, 
                    float *d_oldtemp, 
                    float3* d_vel, 
                    float3* d_oldvel, 
                    float* d_pressure,
                    float3* d_ccvel,
                    float3* d_vorticity,
                    float* d_smokedensity,
                    float* d_oldsmokedensity,
                    float * d_smokeRadiance,
                    int activeBuffer, dim3 Ld, BC bc, dim3 M_in, unsigned int slice) {
    const dim3 gridSize(blocksNeeded(Ld.x, M_in.x), blocksNeeded(Ld.y, M_in.y), 
                        blocksNeeded(Ld.z,M_in.z));
    const size_t smSz = (M_in.x + 2 * RAD)*(M_in.y + 2 * RAD)*(M_in.z + 2 * RAD)*sizeof(float);//shared mem size
    // CFD
    computeVorticity<<<gridSize, M_in>>>(d_vorticity, d_oldvel, d_ccvel);
    HANDLE_ERROR(hipPeekAtLastError());
    velocityKernel<<<gridSize, M_in, smSz>>>(d_oldtemp, d_vel, d_oldvel, d_oldsmokedensity, d_vorticity);
    HANDLE_ERROR(hipPeekAtLastError());
    forceIncompressibility(d_vel, d_pressure);
    tempAdvectionKernel<<<gridSize, M_in, smSz>>>(d_temp, d_oldtemp, d_oldvel, d_oldsmokedensity);
    HANDLE_ERROR(hipPeekAtLastError());
    smokeAdvectionKernel<<<gridSize, M_in, smSz>>>(d_oldtemp, d_oldvel, d_smokedensity, d_oldsmokedensity);
    HANDLE_ERROR(hipPeekAtLastError());
    
    smokeRender(gridSize, d_out, d_smokedensity, d_smokeRadiance);

    //tempKernel<<<gridSize, M_in, smSz>>>(d_temp, bc);
    HANDLE_ERROR(hipDeviceSynchronize());
}

void resetVariables(float* d_temp,
                    float* d_oldtemp,
                    float3* d_vel, 
                    float3* d_oldvel, 
                    float* d_smokedensity,
                    float* d_oldsmokedensity, 
                    dim3 Ld, BC bc, dim3 M_in) {
    const dim3 gridSize( blocksNeeded(Ld.x, M_in.x), blocksNeeded( Ld.y, M_in.y), 
                            blocksNeeded(Ld.z, M_in.z));
    resetKernel<<<gridSize, M_in>>>(d_temp, d_oldtemp, d_vel, d_oldvel, d_smokedensity, d_oldsmokedensity, bc);
    HANDLE_ERROR(hipPeekAtLastError());
    HANDLE_ERROR(hipDeviceSynchronize());
}
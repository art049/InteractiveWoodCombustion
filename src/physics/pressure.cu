#include "hip/hip_runtime.h"
#include "pressure.cuh"

__global__ void prepareSystem(int NFLAT, float3* d_vel, float* d_b, float* d_val, int* d_cind, int * d_rptr) {
    const int k_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int k_y = threadIdx.y + blockDim.y * blockIdx.y;
    const int k_z = threadIdx.z + blockDim.z * blockIdx.z;
    if ((k_x >= dev_Ld[0] ) || (k_y >= dev_Ld[1] ) || (k_z >= dev_Ld[2])) return;
    const int k = flatten(k_x, k_y, k_z);
    //B term
    d_b[k] = d_vel[flatten(k_x+1, k_y, k_z)].x - d_vel[flatten(k_x, k_y, k_z).x] + 
             d_vel[flatten(k_x, k_y+1, k_z)].y - d_vel[flatten(k_x, k_y, k_z).y] + 
             d_vel[flatten(k_x, k_y, k_z+1)].z - d_vel[flatten(k_x, k_y, k_z).z] ;
    d_b[k] /= BLOCK_SIZE * dev_Deltat[0];
    
    // Matrix 
    if(k_x > 0 && k_x < GRID_COUNT - 1 && k_y > 0 && k_y < GRID_COUNT - 1 && k_z > 0 && k_z < GRID_COUNT - 1) {
        const int offset = 7 * flatten(k_x-1, k_y-1, k_z-1, GRID_COUNT-2, GRID_COUNT-2, GRID_COUNT-2);
        d_val [offset    ] = -6; 
        d_cind[offset    ] = k;
        d_val [offset + 1] =  1; 
        d_cind[offset + 1] = flatten(k_x+1, k_y, k_z);
        d_val [offset + 2] =  1; 
        d_cind[offset + 1] = flatten(k_x-1, k_y, k_z);
        d_val [offset + 3] =  1; 
        d_cind[offset + 1] = flatten(k_x, k_y+1, k_z);
        d_val [offset + 4] =  1; 
        d_cind[offset + 1] = flatten(k_x, k_y-1, k_z);
        d_val [offset + 5] =  1; 
        d_cind[offset + 1] = flatten(k_x, k_y, k_z+1);
        d_val [offset + 6] =  1;
        d_cind[offset + 1] = flatten(k_x+1, k_y, k_z-1);
        
        d_rptr[k] 
    }
    //PRESSURE DIRICHLET BOUNDARY CONDITION
    else {

    }


}

void forceIncompressibility(float3 * d_vel, float* d_pressure){
    // TODO: CHOLESKI PREPROCESS
    const int NFLAT = GRID_COUNT * GRID_COUNT * GRID_COUNT;
    const dim3 gridSize(blocksNeeded(GRID_COUNT, M_i.x), 
                        blocksNeeded(GRID_COUNT, M_i.y), 
                        blocksNeeded(GRID_COUNT, M_i.z));

    // CGLS solver config
    float shift = 0;
    float tol = 1e-6;
    int maxit = 20;
    bool quiet = false;
    int m = NFLAT;
    int n = NFLAT;
    int nnz = 7 * NFLAT;
    float *d_val, *d_b;
    int *d_cind, *d_rptr;
    HANDLE_ERROR(hipMalloc(&d_val, (nnz + m) * sizeof(float)));
    d_b = d_val + nnz;
    HANDLE_ERROR(hipMalloc(&d_cind, (nnz + m + 1) * sizeof(int)));
    d_rptr = d_cind + nnz;

    prepareSystem<<<gridSize, M_i>>>(NFLAT, d_vel, d_b, d_val, d_cind, d_rptr);
    HANDLE_ERROR(hipPeekAtLastError());

    int flag = cgls::Solve<float, cgls::CSR>(d_val, d_rptr, d_cind, m, n, nnz, d_b, d_pressure, 
                                             shift, tol, maxit, quiet);
    if (flag != 0)
        printf("[CGLS warning] Flag = %d, Error = %e\n", flag, err);
    HANDLE_ERROR(hipFree(d_val));
    HANDLE_ERROR(hipFree(d_cind));

    HANDLE_ERROR(hipDeviceSynchronize());
}